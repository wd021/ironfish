#include <stdarg.h>
#include <stddef.h>
#include <setjmp.h>
#include <hip/hip_runtime_api.h>
// #include <unistd.h>
#include <string.h>
#include <chrono>
#include <thread>
#include <ctime>
#include <assert.h>
//#include <windows.h>
// #include <cmocka.h>
#include "./blake3/blake3.cu"

 //!!nvcc -c  test.cu --compiler-options -fPIC

#if !defined(ssize_t)
typedef long ssize_t;
#endif

#define bzero(b, len) (memset((b), '\0', (len)), (void) 0)

typedef struct blob_t {
    uint8_t *blob;
    ssize_t len;
} blob_t;

void free_blob(blob_t *blob)
{
    free(blob->blob);
}

char *bytes_to_hex(uint8_t *bytes, ssize_t len)
{
    ssize_t hex_len = 2 * len + 1;
    char *hex_string = (char *)malloc(hex_len);
    memset(hex_string, 0, hex_len);

    uint8_t *byte_cursor = bytes;
    char *hex_cursor = hex_string;
    ssize_t count = 0;
    while (count < len) {
        sprintf(hex_cursor, "%02x", *byte_cursor);
        byte_cursor++;
        count++;
        hex_cursor += 2;
    }

    return hex_string;
}

char hex_to_byte(char hex)
{
    if (hex >= '0' && hex <= '9') {
        return hex - '0';
    } else if (hex >= 'a' && hex <= 'f') {
        return hex - 'a' + 10;
    } else {
        exit(1);
    }
}

void hex_to_bytes(const char *hex_data, blob_t *buf)
{
    // printf("789987");
    size_t hex_len = strlen(hex_data);
    assert(hex_len % 2 == 0);

    buf->len = hex_len / 2;
    buf->blob = (uint8_t *)malloc(buf->len);
    memset(buf->blob, 0, buf->len);

    for (size_t pos = 0; pos < hex_len; pos += 2) {
        char left = hex_to_byte(hex_data[pos]);
        char right = hex_to_byte(hex_data[pos + 1]);
        buf->blob[pos / 2] = (left << 4) + right;
    }
}

char* mine(const char *header_string, const char *target_string){
    hipProfilerStart();
    blob_t blob;
    hex_to_bytes(
            header_string,
            &blob);

    blob_t target;
    hex_to_bytes(target_string, &target);

    inline_blake::blake3_hasher *hasher;
    inline_blake::blake3_hasher *device_hasher1;
    TRY(hipHostMalloc(&hasher, sizeof(inline_blake::blake3_hasher)));
    TRY(hipMalloc(&device_hasher1, sizeof(inline_blake::blake3_hasher)));

    bzero(hasher->buf, BLAKE3_BUF_CAP);
    hipMemcpy(hasher->buf, blob.blob, sizeof(BLAKE3_BUF_CAP), hipMemcpyHostToDevice);
    hipMemcpy(hasher->target, target.blob, sizeof(32), hipMemcpyHostToDevice);
    memcpy(hasher->target, target.blob, target.len);
    memcpy(hasher->buf, blob.blob, blob.len);
    hasher->from_group = 2;
    hasher->to_group = 2;

    hipStream_t stream;
    TRY(hipStreamCreate(&stream));
    TRY(hipMemcpyAsync(device_hasher1, hasher, sizeof(inline_blake::blake3_hasher), hipMemcpyHostToDevice, stream));
    inline_blake::blake3_hasher_mine<<<92, 256, 0, stream>>>(device_hasher1);
    TRY(hipStreamSynchronize(stream));
    hipDeviceSynchronize();

    TRY(hipMemcpy(hasher, device_hasher1, sizeof(inline_blake::blake3_hasher), hipMemcpyDeviceToHost));
    char *hash_string1 = bytes_to_hex(hasher->hash, 32);
    char *buf_string1 = bytes_to_hex(hasher->buf, 208);
    printf("good: %d\n", hasher->found_good_hash);
    printf("nonce: %d\n", hasher->buf[0]);
    printf("buf: %d\n", hasher->buf);
    printf("count: %d\n", hasher->hash_count);
    printf("%s\n", hash_string1); 
    printf("%s\n", buf_string1);
    hipProfilerStop();
    return buf_string1;
}


// int main(void) {
//     hipProfilerStart();
//     blob_t blob;
//     hex_to_bytes(
//             "0000000000000000cf0e020000000000000000000002aaced825176dd9db0701c995760a03a1f42c69b63b4b7d4090b0ff7f32477b07a0cc3c89d6f6335433def2d95ff91be838ae47212ba43794901bb0ce220200000000f6ee7f75663920ae6d8617379629d5130323e6e20c5e19cb5606c71bb97ed7e668d5130100000000000000000007b87e00ba71e3b4a9a27d79dad30a55297da63550092644b289502c8efe8f82010000000000007736f4a168656e7461693800000000000000000000000000000000000000000000000000",
//             &blob);

//     blob_t target;
//     hex_to_bytes("0000000fffffffffffffffffffffffff", &target);

//     inline_blake::blake3_hasher *hasher;
//     inline_blake::blake3_hasher *device_hasher1;
//     TRY(hipHostMalloc(&hasher, sizeof(inline_blake::blake3_hasher)));
//     TRY(hipMalloc(&device_hasher1, sizeof(inline_blake::blake3_hasher)));

//     bzero(hasher->buf, BLAKE3_BUF_CAP);
//     hipMemcpy(hasher->buf, blob.blob, sizeof(BLAKE3_BUF_CAP), hipMemcpyHostToDevice);
//     hipMemcpy(hasher->target, target.blob, sizeof(32), hipMemcpyHostToDevice);
//     memcpy(hasher->target, target.blob, target.len);
//     memcpy(hasher->buf, blob.blob, blob.len);
//     hasher->from_group = 2;
//     hasher->to_group = 2;

//     hipStream_t stream;
//     TRY(hipStreamCreate(&stream));
//     TRY(hipMemcpyAsync(device_hasher1, hasher, sizeof(inline_blake::blake3_hasher), hipMemcpyHostToDevice, stream));
//     inline_blake::blake3_hasher_mine<<<92, 256, 0, stream>>>(device_hasher1);
//     TRY(hipStreamSynchronize(stream));
//     hipDeviceSynchronize();

//     TRY(hipMemcpy(hasher, device_hasher1, sizeof(inline_blake::blake3_hasher), hipMemcpyDeviceToHost));
//     char *hash_string1 = bytes_to_hex(hasher->hash, 32);
//     char *buf_string1 = bytes_to_hex(hasher->buf, 208);
//     printf("good: %d\n", hasher->found_good_hash);
//     printf("nonce: %d\n", hasher->buf[0]);
//     printf("buf: %d\n", hasher->buf);
//     printf("count: %d\n", hasher->hash_count);
//     printf("%s\n", hash_string1); 
//     printf("%s\n", buf_string1);
//     hipProfilerStop();
// }